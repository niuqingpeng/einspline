#include <stdio.h>
#include <hip/hip_runtime.h>




extern "C" void cudaInit(size_t sizeA);
extern "C" void cudaFinalize();
extern "C" void putGPU(void* h_A, size_t sizeA);
extern "C" void getGPU(void* h_A, size_t sizeA);

void* d_A;


void cudaInit(size_t sizeA){

	//allocate memory on device
	hipMalloc( (void**) &d_A, sizeA);
}

void putGPU(void* h_A, size_t sizeA){

	//copy host data from argument to device
	hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
}


void getGPU(void* h_A, size_t sizeA){
	
	//copy data from device to argument array
	hipMemcpy(h_A, d_A, sizeA, hipMemcpyDeviceToHost);
}

void cudaFinalize(){

	//free device memory
	hipFree(d_A);
}


